#include "hip/hip_runtime.h"
#include <fstream>
#include "hip/hip_runtime.h"
#include "kernel.hpp"
#include "cuda_error.hpp"
#include <iostream>

using namespace cu_geodesic;

// copy certain "constants" to gpu memory at compile time
__device__ const static double d_max = std::numeric_limits<double>::max();
__device__ const static double inf = std::numeric_limits<double>::infinity();
__device__ const static double reasonable_delta = 1e-3;

// macros around cuda error logger
#define CUDA_SAFE_NO_RET(call) {                                               \
    hipError_t err = call;                                                    \
    if (err != hipSuccess) {                                                  \
      cuda_log_error(err, #call, __FILE__, __LINE__);                          \
    } \
  }

#define CUDA_SAFE(call)                                                        \
  {                                                                            \
    hipError_t err = call;                                                    \
    if (err != hipSuccess) {                                                  \
      cuda_log_error(err, #call, __FILE__, __LINE__);                          \
      return std::nullopt; \
    } \
  }

#define CUDA_KERNEL(call) {                                                    \
      call;                                                                    \
      hipError_t err = hipGetLastError();                                    \
      if(err != hipSuccess) {                                                 \
        cuda_log_error(err, #call, __FILE__, __LINE__);                        \
        return std::nullopt;                                                   \
    }                                                                          \
  }

// result type on GPU
struct cu_res {
  double distance;
  point pt;
};

// std::optional<point>, but less fancy and very GPU friendly
struct intersection {
  bool is_intersection;
  point pt;
};

// wrapper around GPU pointers for arrays
template <typename T>
struct cu_arr {
  // array size
  size_t size;
  // array data buffer
  T *data;

  // allocation is only possible on the host
  __host__ static cu_arr<T> alloc(size_t size) {
    cu_arr<T> res { .size = size, .data = nullptr };
    CUDA_SAFE_NO_RET(hipMallocManaged(&res.data, sizeof(T) * size))
    return res;
  }

  // copies data into the buffer, from anywhere, is only possible on the host
  __host__ static cu_arr<T> copy(size_t size, T *buf) {
    cu_arr<T> res = alloc(size);
    CUDA_SAFE_NO_RET(hipMemcpy(res.data, buf, sizeof(T) * size, hipMemcpyKind::hipMemcpyDefault))
    return res;
  }

  // cleanup is only possible on the host (frees the memory)
  __host__ void cleanup() {
    if(data != nullptr) {
      CUDA_SAFE_NO_RET(hipFree(data))
      data = nullptr;
      size = 0;
    }
  }
};

// GPU representation of the polygon (uses cu_arr instead of std::vector)
struct cu_poly {
  // the status of a point (outside, inside, in a hole, on any edge)
  enum struct status { OUTSIDE, POLYGON, HOLE, ON_EDGE };

  // getting the status of a point is only possible on the GPU
  [[nodiscard]] __device__ status status_for(const point &p) const;
  // checking if a line segment intersects any edge is only possible on the GPU
  [[nodiscard]] __device__ bool intersects_any(const segment &s) const;

  // the segments in the boundary
  cu_arr<segment> boundary;
  // the holes (as lists of segments)
  cu_arr<cu_arr<segment>> holes;
};

// GPU representation of the adjacency graph for the vertices in the polygon
class cu_graph {
public:
  // iterator over the neighbors of a vertex in the graph
  struct iterator {
    // current index of the point
    size_t idx;
    // point whose neighbors we're iterating over
    const size_t source;
    // graph we're using
    const cu_graph &graph;

    // gets the next neighbor (if any) - only mutates, doesn't return anything
    __device__ void operator++() {
      idx++; // increment first
      while(idx < graph.size()) {
        if(!isinf(graph.distance_between(source, idx)) && idx != source) {
          return;
        }
        idx++;
      }
    }

    // checks if we're still in the graph's bounds
    __device__ operator bool() const {
      return idx < graph.size();
    }
  };

  // constructing a graph is only possible on the host
  __host__ cu_graph(const cu_poly *polygon) : polygon{polygon} {
    vertex_count = polygon->boundary.size;
    for(size_t idx = 0; idx < polygon->holes.size; idx++) {
      vertex_count += polygon->holes.data[idx].size;
    }
    // we are grossly over-allocating this, but let me be for now
    hipMallocManaged(&distances, vertex_count * vertex_count * sizeof(double));
  }

  // cleaning up the graph is only possible on the host (aka free)
  __host__ void clean() {
    if(distances != nullptr)
      hipFree(distances);
    distances = nullptr;
  }

  // gets a pointer to the adjacency matrix
  __host__ const double *adjacencies() const { return distances; }

  // looks for a pont in the polygon (same as in the CPU polygon structure)
  __device__ const point &vertex(size_t idx) const {
    if(idx < polygon->boundary.size) {
      return polygon->boundary.data[idx].begin;
    }
    idx -= polygon->boundary.size;
    for(size_t hole = 0; hole < polygon->holes.size; hole++) {
      if(idx < polygon->holes.data[hole].size) {
        return polygon->holes.data[hole].data[idx].begin;
      }
      idx -= polygon->holes.data[hole].size;
    }
  }

  // gets a modifiable reference in the adjacency matrix
  __device__ double &distance_between(size_t idx1, size_t idx2) {
    return distances[idx1 * vertex_count + idx2];
  }

  // gets a non-modifiable reference in the adjacency matrix
  __device__ const double &distance_between(size_t idx1, size_t idx2) const {
    return distances[idx1 * vertex_count + idx2];
  }

  // gets an iterator for the neighbors of a vertex (ready at the first one)
  __device__ iterator neighbors_for(size_t idx) {
    size_t first = 0;
    while(first < size() && isinf(distance_between(idx, first))) first++;
    return {
        .idx = first,
        .source = idx,
        .graph = *this,
    };
  }

  // gets the polygon (unmodifiable)
  __device__ const cu_poly &poly() const {
    return *polygon;
  }

  // gets the amount of vertices
  __host__ __device__ size_t size() const {
    return vertex_count;
  }

private:
  // constant pointer to the polygon (references usually don't behave with CUDA)
  const cu_poly *polygon;
  // the amount of vertices (aka sqrt(amount of elements in distances))
  size_t vertex_count;
  // the adjacency matrix
  double *distances = nullptr;
};

// a simple GPU priority queue (using as little as possible allocations)
class a_star_queue {
public:
  // element type (not to be confused with the node type)
  struct elem {
    // estimates the total distance/cost (backward + forward)
    [[nodiscard]] __device__ double estimate() const { return until_now + forward; }
    // backward cost
    double until_now;
    // forward cost
    double forward;
    // vertex index
    size_t value;
  };

  // is the queue empty?
  __device__ bool empty() const {
    return _used == 0;
  }

  // enqueues a new node (extending the array if needed)
  // in that case the array is extended, removes all gravestones
  __device__ void enqueue(const elem &e) {
    if(_size + 1 >= _cap) {
      node *upd = (node *)malloc((size_t)((float)_cap * 1.5) * sizeof(node));
      _cap = (size_t)((float)_cap * 1.5);
      size_t upd_size = 0;
      for(size_t i = 0; i < _size; i++) {
        if(!data[i].is_gravestone) upd[upd_size] = data[i];
        upd_size++;
      }

      free(data);
      data = upd;
      _size = upd_size;
    }

    data[_size] = { .val = e, .is_gravestone = false };
    _size++;
    _used++;
  }

  // finds the element with the highest priority (the lowest estimated distance)
  // and removes it (replace it with a gravestone)
  __device__ elem dequeue() {
    double lowest = d_max;
    size_t lowest_idx = 0;
    for(size_t i = 0; i < _size; i++) {
      if(!data[i].is_gravestone && data[i].val.estimate() < lowest) {
        lowest = data[i].val.estimate();
        lowest_idx = i;
      }
    }

    data[lowest_idx].is_gravestone = true;
    _used--;
    return data[lowest_idx].val;
  }

  // frees all data
  __device__ void cleanup() {
    if(data != nullptr) {
      free(data);
      data = nullptr;
      _size = 0;
      _used = 0;
      _cap = 0;
    }
  }

  // looks if a vertex is in the queue & returns queue buffer index
  [[nodiscard]] __device__ size_t find(size_t v) const {
    for(size_t i = 0; i < _size; i++) {
      if(!data[i].is_gravestone && v == data[i].val.value) {
        return i;
      }
    }
    return _size;
  }

  // gets a ref to an element by queue buffer index
  __device__ elem &get(size_t idx) {
    return data[idx].val;
  }

  // gets the size of the array (not the used slots, not the capacity)
  [[nodiscard]] __device__ size_t size() const { return _size; }

private:
  // a node in the queue (less fancy std::optional<elem>)
  struct node {
    // the value in the node
    elem val;
    // is this node a gravestone?
    bool is_gravestone;
  };

  // the actual buffer
  node *data = (node *)malloc(64 * sizeof(node));
  // the next index to be used
  size_t _size = 0;
  // the amount of actually used slots (_size - amount of gravestones)
  size_t _used = 0;
  // capacity of the container
  size_t _cap = 64;
};

// is x between b1 and b2 (taking into accord the ordering between b1 and b2)
__device__ inline bool between(double b1, double x, double b2) {
  return (b1 < b2) ? (b1 <= x && x <= b2) : (b1 >= x && x >= b2);
}

// are these points exactly the same?
__device__ inline bool point_same(const point &p1, const point &p2) {
  return p1.x == p2.x && p1.y == p2.y;
}

// computes the distance between two points
__device__ double distance(point p1, point p2) {
  double tmp1 = p1.x - p2.x;
  double tmp2 = p1.y - p2.y;
  return sqrt(tmp1 * tmp1 + tmp2 * tmp2);
}

// checks if two line segments intersect
__device__ intersection intersects(const segment &s1, const segment &s2) {
  // check for common end points
  if(point_same(s1.begin, s2.begin) || point_same(s1.begin, s2.end)
      || point_same(s1.end, s2.begin) || point_same(s1.end, s2.end)) {
    return {.is_intersection = false, .pt = {}};
  }
  double cross_x = (s2.b - s1.b) / (s1.a - s2.a);
  bool on_s1 = between(s1.begin.x, cross_x, s1.end.x);
  bool on_s2 = between(s2.begin.x, cross_x, s2.end.x);
  if(on_s1 && on_s2) {
    point cross{
      .x = cross_x,
      .y = s1.a * cross_x + s1.b
    };
    return {.is_intersection = true,
            .pt = cross};
  }
  else {
    return {.is_intersection = false, .pt = {}};
  }
}

// checks if a point is inside a single polygon
__device__ bool inside_single(point p, const cu_arr<segment> &polygon) {
  // adapted from https://en.wikipedia.org/wiki/Even%E2%80%93odd_rule
  bool res = false;
  point prev = polygon.data[polygon.size - 1].begin;
  for (size_t i = 0; i < polygon.size; i++) {
    point curr = polygon.data[i].begin;
    if (p.x == curr.x && p.y == curr.y) {
      return true;
    }
    if ((curr.y > p.y) != (prev.y > p.y)) {
      double slope = (p.x - curr.x) * (prev.y - curr.y) -
                     (prev.x - curr.x) * (p.y - curr.y);
      if ((slope < 0) != (prev.y < curr.y)) {
        res = !res;
      }
    }
    prev = curr;
  }
  return res;
}

// is this point on the given line segment?
__device__ bool point_on(const point &p, const segment &s) {
  if(!between(s.begin.x, p.x, s.end.x)) return false;
  double expected_y = s.a * p.x + s.b;
  return abs(expected_y - p.y) <= 1e-9;
}

// implementation for a member function (relies on previous free functions)
__device__ cu_poly::status cu_poly::status_for(const point &p) const {
  for(size_t i = 0; i < boundary.size; i++) {
    if(point_on(p, boundary.data[i])) return status::ON_EDGE;
  }

  for(size_t i = 0; i < holes.size; i++) {
    for(size_t j = 0; j < holes.data[i].size; j++) {
      if(point_on(p, holes.data[i].data[j])) return status::ON_EDGE;
    }
  }

  if(!inside_single(p, boundary)) { return status::OUTSIDE; }

  for(size_t i = 0; i < holes.size; i++) {
    if(inside_single(p, holes.data[i])) {
      return status::HOLE;
    }
  }

  return status::POLYGON;
}

// implementation for a member function (relies on previous free functions)
__device__ bool cu_poly::intersects_any(const segment &s) const {
  for(size_t i = 0; i < boundary.size; i++) {
    if(intersects(s, boundary.data[i]).is_intersection) return true;
  }

  for(size_t i = 0; i < holes.size; i++) {
    for(size_t j = 0; j < holes.data[i].size; j++) {
      if(intersects(s, holes.data[i].data[j]).is_intersection) return true;
    }
  }

  return false;
}

// computes a line segment from two points
__device__ segment compute_segment(const point &p1, const point &p2) {
  double tmp = (p2.y - p1.y) / (p2.x - p1.x);
  return {
    .begin = p1,
    .end = p2,
    .a = tmp,
    .b = p1.y - tmp * p1.x
  };
}

// interpolate delta units from the starting point of the segment
__device__ point interpolate_from_begin(const segment &s, double delta) {
  if(abs(s.begin.x - s.end.x) < 1e-6) {
    // almost-vertical case; "rotate" 90 degrees, interpolate
    double a = (s.end.x - s.begin.x) / (s.end.y - s.begin.y);
    double b = s.begin.x - a * s.begin.y;
    // correct interpolation direction
    double mod = s.begin.y < s.end.y ? delta : -delta;
    return {
      .x = a * (s.begin.y + mod) + b,
      .y = s.begin.y + mod
    };
  }
  else if(s.begin.x < s.end.x) {
    return {
      .x = s.begin.x + delta,
      .y = s.a * (s.begin.x + delta) + s.b
    };
  }
  else if(s.begin.x > s.end.x) {
    return {
      .x = s.begin.x - delta,
      .y = s.a * (s.begin.x - delta) + s.b
    };
  }
}

// checks if an edge is viable (entirely inside the polygon, not inside a hole
__device__ bool edge_viable(segment s, const cu_graph &graph) {
  point interpolate = interpolate_from_begin(s, reasonable_delta);
  auto inside = graph.poly().status_for(interpolate);
  // check if edge can be entirely within polygon
  if(inside == cu_poly::status::ON_EDGE) {
    // edge is an edge of the polygon
    return true;
  }
  else if(inside == cu_poly::status::POLYGON) {
    // edge starts inside polygon
    // check for any intersection
    if(!graph.poly().intersects_any(s)) {
      // no intersections, add distance
      return true;
    }
  }

  return false;
}

// computes the graph's adjacency matrix
__global__ void compute_graph(cu_graph graph, size_t idx_offset) {
  uint idx = threadIdx.x + blockDim.x * blockIdx.x + idx_offset;
  if(idx >= graph.size()) return; // culling

  graph.distance_between(idx, idx) = 0.0; // should be obvious
  point start = graph.vertex(idx);
  for(size_t other = idx + 1; other < graph.size(); other++) {
    point partner = graph.vertex(other);
    segment seg = compute_segment(start, partner);

    if(edge_viable(seg, graph)) {
      double dist = distance(start, partner);
      graph.distance_between(idx, other) = dist;
      graph.distance_between(other, idx) = dist;
    }
    else {
      // set distance to infinity to make sure
      graph.distance_between(idx, other) = inf;
      graph.distance_between(other, idx) = inf;
    }
  }
}

// computes the geodesic distance array (for each vertex to the start)
__global__ void compute_geodesic_distance(uint max_idx, point start, cu_arr<double> res, cu_graph graph, size_t idx_offset) {
  uint idx = threadIdx.x + blockDim.x * blockIdx.x + idx_offset;
  if(idx >= max_idx) return; // culling

  // check viability of point, project if necessary
  // points are always viable as they are graph vertices
  point target = graph.vertex(idx);

  // check for direct route
  if(edge_viable(compute_segment(start, target), graph)) {
    res.data[idx] = distance(start, target);
    return; // done
  }

  a_star_queue queue;
  for(size_t i = 0; i < graph.size(); i++) {
    auto v = graph.vertex(i);
    auto e = compute_segment(v, start);
    if(edge_viable(e, graph)) {
      double d = distance(v, start);
      double f = distance(v, target);
      queue.enqueue({
          .until_now = d,
          .forward = f,
          .value = i
      });
    }
  }

  // A*
  while(!queue.empty()) {
    auto next = queue.dequeue();
    auto v = graph.vertex(next.value);
    if(next.forward == 0.0) {
      // found end: distance squared from any point to target = 0? point = target
      res.data[idx] = next.until_now;
      break; // done!
    }
    auto iter = graph.neighbors_for(next.value);
    while(iter) {
      auto v_ = graph.vertex(iter.idx);
      double tentative = next.until_now + graph.distance_between(next.value, iter.idx);
      size_t queue_idx = queue.find(iter.idx);
      if(queue_idx == queue.size()) {
        // not yet in queue
        queue.enqueue({
            .until_now = tentative,
            .forward = distance(graph.vertex(iter.idx), target),
            .value = iter.idx
        });
      }
      else {
        // in queue
        auto &elem = queue.get(queue_idx);
        auto _v_ = graph.vertex(elem.value);
        if(tentative < elem.until_now) {
          elem.until_now = tentative;
        }
      }

      ++iter;
    }
  }

  // clean up
  queue.cleanup();
}

// computes the distance to the end point using the geodesic distance array
__global__ void compute_to_point(uint max_idx, point start, cu_graph graph, point min, size_t per_row, double granularity, cu_arr<double> point_dist, cu_res *res, size_t idx_offset) {
  uint idx = threadIdx.x + blockDim.x * blockIdx.x + idx_offset;
  if(idx > max_idx) return; // out of range

  // compute target
  uint row_idx = idx / per_row;
  uint col_idx = idx % per_row;
  point target {
      .x = col_idx * granularity + min.x,
      .y = row_idx * granularity + min.y
  };
  res[idx].pt = target;
  auto in = graph.poly().status_for(target);
  if(in != cu_poly::status::POLYGON && in != cu_poly::status::ON_EDGE) {
    res[idx].distance = -1.0; // point is out of bounds
    return;
  }

  // check straight edge
  segment straight = compute_segment(start, target);
  if(edge_viable(straight, graph)) {
    res[idx].distance = distance(start, target);
    return;
  }

  // check each other vertex
  double min_dist = inf;
  for(size_t vert = 0; vert < graph.size(); vert++) {
    point vertex = graph.vertex(vert);
    segment between = compute_segment(vertex, target);
    if(edge_viable(between, graph)) {
      // Euclidean distance + backward distance from vertex
      double dist = distance(vertex, target) + point_dist.data[vert];
      if(dist < min_dist) {
        min_dist = dist;
      }
    }
  }

  // no path?
  res[idx].distance = (isinf(min_dist)) ? -1.0 : min_dist;
}

// implementation of the kernel wrapper
__host__ std::optional<result> kernel::operator()(double granularity, size_t core_cnt) {
  // alloc buffers & copy data
  std::cout << "[KERNEL]: allocating GPU memory and copying data..." << std::endl;
  cu_poly *polygon;
  cu_poly _polygon {
    .boundary = cu_arr<segment>::copy(data.boundary.size(), data.boundary.data()),
    .holes = cu_arr<cu_arr<segment>>::alloc(data.holes.size())
  };
  for(size_t i = 0; i < data.holes.size(); i++) {
    _polygon.holes.data[i] = cu_arr<segment>::copy(data.holes[i].size(), data.holes[i].data());
  }
  CUDA_SAFE(hipMallocManaged(&polygon, sizeof(cu_poly)))
  CUDA_SAFE(hipMemcpy(polygon, &_polygon, sizeof(cu_poly), hipMemcpyKind::hipMemcpyDefault))
  cu_graph graph(polygon);

  std::cout << "[KERNEL]: step one: starting GPU graph generation (attempting to use " << graph.size() << " CUDA threads)..." << std::endl;
  for(size_t i = 0; i < graph.size(); i += core_cnt) {
    CUDA_KERNEL((compute_graph<<<1, core_cnt>>>(graph, i)))
    CUDA_SAFE(hipDeviceSynchronize()) // wait till cores are done
  }
  std::cout << "[KERNEL]: step one finished. GPU graph is generated." << std::endl;

  std::cout << "[KERNEL]: outputting GPU graph adjacencies..." << std::endl;
  {
    // sub scope for cleanup
    std::vector<double> adj(graph.size() * graph.size());
    CUDA_SAFE(hipMemcpy(adj.data(), graph.adjacencies(),
                         graph.size() * graph.size() * sizeof(double),
                         hipMemcpyKind::hipMemcpyDefault))
    std::ofstream adj_dump("/tmp/adjacencies.txt");
    if (adj_dump.is_open()) {
      adj_dump << "from\\to";
      for (size_t i = 0; i < graph.size(); i++) {
        point p = data.vertex(i);
        adj_dump << "\t(" << p.x << "," << p.y << ")";
      }
      adj_dump << std::endl;
      for (size_t i = 0; i < graph.size(); i++) {
        point p2 = data.vertex(i);
        adj_dump << "(" << p2.x << "," << p2.y << ")";
        for (size_t j = 0; j < graph.size(); j++) {
          adj_dump << "\t" << adj[i * graph.size() + j];
        }
        adj_dump << std::endl;
      }
    }
  }
  std::cout << "[KERNEL]: GPU graph adjacencies outputted to /tmp/adjacencies.txt." << std::endl;

  // new second step: compute distance from start to each graph vertex
  std::cout << "[KERNEL]: step two: calculate geodesic distances from each vertex to the start..." << std::endl;
  cu_arr<double> to_vertices { .size = graph.size(), .data = nullptr };
  CUDA_SAFE(hipMallocManaged(&to_vertices.data, graph.size() * sizeof(double)))
  for(size_t i = 0; i < graph.size(); i += core_cnt) {
    CUDA_KERNEL((compute_geodesic_distance<<<1, core_cnt>>>(graph.size(), start, to_vertices, graph, i)))
    CUDA_SAFE(hipDeviceSynchronize()) // wait till cores are done
  }
  std::cout << "[KERNEL]: step two finished." << std::endl;
  std::cout << "[KERNEL]: cleaning up graph..." << std::endl;
  graph.clean();
  std::cout << "[KERNEL]: outputting geodesic distances to vertices..." << std::endl;
  {
    // sub scope for cleanup
    std::ofstream strm("/tmp/geodesic.csv");
    if(strm.is_open()) {
      std::vector<double> tmp(graph.size());
      strm << "vertex x,vertex y,distance from start" << std::endl;
      CUDA_SAFE(hipMemcpy(tmp.data(), to_vertices.data, graph.size() * sizeof(double), hipMemcpyKind::hipMemcpyDefault))
      for(size_t i = 0; i < tmp.size(); i++) {
        const auto p = data.vertex(i);
        strm << p.x << "," << p.y << "," << tmp[i] << std::endl;
      }
    }
  }
  std::cout << "[KERNEL]: geodesic distances outputted to /tmp/geodesic.csv." << std::endl;

  bounding_box box = start_box;
  std::cout << "[KERNEL]: step three: calculate geodesic distances to points..." << std::endl;
  auto required_x_steps = (size_t)std::ceil((start_box.max.x - start_box.min.x) / granularity);
  auto required_y_steps = (size_t)std::ceil((start_box.max.y - start_box.min.y) / granularity);
  cu_res *res;
  CUDA_SAFE(hipMallocManaged(&res, required_x_steps * required_y_steps * sizeof(cu_res)))
  std::cout << "[KERNEL]:   requiring " << required_x_steps << " steps in x-dimension, and "
            << required_y_steps << " in y-dimension; " << required_x_steps*required_y_steps
            << " steps in total" << std::endl;

  size_t total_steps = required_x_steps * required_y_steps;
  result res_cpu{
      .distances = std::vector<result::matrix_pt>(total_steps),
      .farthest = {},
      .contained = start_box
  };
  // run kernel
  for(size_t i = 0; i < total_steps; i += core_cnt) {
    CUDA_KERNEL((compute_to_point<<<1, core_cnt>>>(
        total_steps, start, graph, start_box.min, required_x_steps,
        granularity, to_vertices, res, i
    )))
    CUDA_SAFE(hipDeviceSynchronize()) // wait for threads to finish
    std::cout << "\r[KERNEL]:   ran " << i << "/" << total_steps << " ("
              << (int)((double)i / (double)total_steps * 100) << "%) steps" << std::flush;
  }
  std::cout << std::endl << "[KERNEL]: step three finished." << std::endl;

  std::cout << "[KERNEL]: step four: copy results to CPU..." << std::endl;
  std::vector<cu_res> distances_cpu(total_steps);
  CUDA_SAFE(hipMemcpy(distances_cpu.data(), res, distances_cpu.size() * sizeof(cu_res), hipMemcpyKind::hipMemcpyDefault))
  double farthest = 0.0;
  for(size_t i = 0; i < distances_cpu.size(); i++) {
    const auto &p = distances_cpu[i];
    res_cpu.distances[i] = {
        .distance = distances_cpu[i].distance,
        .pt = distances_cpu[i].pt
    };
    if(p.distance > farthest) {
      farthest = p.distance;
      res_cpu.farthest = p.pt;
    }

    if(i % 1000 == 0) {
      std::cout << "\r[KERNEL]:   copied/checked " << i << "/"
                << distances_cpu.size() << " ("
                << 100 * i / distances_cpu.size() << "%) values." << std::flush;
    }
  }
  res_cpu.contained = box;
  std::cout << "\n[KERNEL]: step four finished. Farthest point is (" << res_cpu.farthest.x
            << ", " << res_cpu.farthest.y << "); distance " << farthest << "." << std::endl;

  std::cout << "[KERNEL]: computations done running. Cleaning up..." << std::endl;
  to_vertices.cleanup();
  _polygon.boundary.cleanup();
  for(size_t i = 0; i < _polygon.holes.size; i++) {
    _polygon.holes.data[i].cleanup();
  }
  _polygon.holes.cleanup();
  CUDA_SAFE(hipFree(polygon))
  CUDA_SAFE(hipFree(res))

  std::cout << "[KERNEL]: kernel finished running." << std::endl;
  return res_cpu;
}
