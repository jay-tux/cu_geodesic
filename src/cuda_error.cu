#include "cuda_error.hpp"
#include "hip/hip_runtime.h"
#include <iostream>

void cu_geodesic::cuda_log_error(hipError_t err, const char *call,
                                 const char *file, size_t line) {
  std::cerr << "[ERROR]: " << file << " on line " << line
            << "; while executing `" << call << "': " << hipGetErrorString(err)
            << std::endl;
  hipGetLastError(); // reset error to no error
}