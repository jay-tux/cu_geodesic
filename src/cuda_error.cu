#include <iostream>
#include "hip/hip_runtime.h"
#include "cuda_error.hpp"

void cudijkstra::cuda_log_error(hipError_t err, const char *call, const char *file, size_t line) {
  std::cerr << "[ERROR]: " << file << " on line " << line
            << "; while executing `" << call << "': " << hipGetErrorString(err)
            << std::endl;
  hipGetLastError(); // reset error to no error
}